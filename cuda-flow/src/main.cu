#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// Kernel to raise each element to a power (e.g., square each element)
__global__ void powerKernel(float* data, int size, float power)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) { data[idx] = powf(data[idx], power); }
}

// Function to check CUDA error
void checkCudaError(hipError_t err, const char* msg)
{
  if (err != hipSuccess) {
    std::cerr << "Error: " << msg << " (" << hipGetErrorString(err) << ")" << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main()
{
  // Parameters
  const int arraySize = 1 << 25;
  const float power = 2.0f;          // Square each element
  const float searchValue = 1024.0f; // Value to search after power operation
  const int N = 10;                  // Number of repetitions

  // Allocate and initialize data on CPU
  int i = 0;
  std::vector<float> hostData(arraySize);
  std::generate(hostData.begin(), hostData.end(), [&i]() { return i++; });

  // Allocate memory on GPU
  float* deviceData;
  checkCudaError(hipMalloc(&deviceData, arraySize * sizeof(float)), "Failed to allocate device memory");

  // Timing variables for accumulation
  float totalCpuToGpuTime = 0.0f;
  float totalGpuComputeTime = 0.0f;
  float totalGpuToCpuTime = 0.0f;
  float totalCpuComputeTime = 0.0f;

  for (int i = 0; i < N; ++i) {
    // Copy data to GPU
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaError(
      hipMemcpy(deviceData, hostData.data(), arraySize * sizeof(float), hipMemcpyHostToDevice),
      "Failed to copy data to GPU");
    auto end = std::chrono::high_resolution_clock::now();
    float cpuToGpuTime = std::chrono::duration<float, std::milli>(end - start).count();
    totalCpuToGpuTime += cpuToGpuTime;
    std::cout << "Iteration " << i + 1 << " - CPU to GPU transfer time: " << cpuToGpuTime << " ms\n";

    // Launch kernel to perform power operation
    int threadsPerBlock = 256;
    int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    start = std::chrono::high_resolution_clock::now();
    powerKernel<<<blocksPerGrid, threadsPerBlock>>>(deviceData, arraySize, power);
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");
    end = std::chrono::high_resolution_clock::now();
    float gpuComputeTime = std::chrono::duration<float, std::milli>(end - start).count();
    totalGpuComputeTime += gpuComputeTime;
    std::cout << "Iteration " << i + 1 << " - GPU computation time: " << gpuComputeTime << " ms\n";

    // Copy result back to CPU
    start = std::chrono::high_resolution_clock::now();
    checkCudaError(
      hipMemcpy(hostData.data(), deviceData, arraySize * sizeof(float), hipMemcpyDeviceToHost),
      "Failed to copy data to CPU");
    end = std::chrono::high_resolution_clock::now();
    float gpuToCpuTime = std::chrono::duration<float, std::milli>(end - start).count();
    totalGpuToCpuTime += gpuToCpuTime;
    std::cout << "Iteration " << i + 1 << " - GPU to CPU transfer time: " << gpuToCpuTime << " ms\n";

    // Perform binary search on CPU
    start = std::chrono::high_resolution_clock::now();
    bool found = std::binary_search(hostData.begin(), hostData.end(), searchValue);
    end = std::chrono::high_resolution_clock::now();
    float cpuComputeTime = std::chrono::duration<float, std::milli>(end - start).count();
    totalCpuComputeTime += cpuComputeTime;
    std::cout << "Iteration " << i + 1 << " - CPU binary search time: " << cpuComputeTime << " ms\n";
    std::cout << "Iteration " << i + 1 << " - Value " << (found ? "found" : "not found") << " in the array.\n";
  }

  // Calculate average times
  float avgCpuToGpuTime = totalCpuToGpuTime / N;
  float avgGpuComputeTime = totalGpuComputeTime / N;
  float avgGpuToCpuTime = totalGpuToCpuTime / N;
  float avgCpuComputeTime = totalCpuComputeTime / N;

  // Display the average times and compute transfer/computation overheads
  std::cout << "\n=== Performance Summary ===\n";
  std::cout << "Average CPU to GPU transfer time: " << avgCpuToGpuTime << " ms\n";
  std::cout << "Average GPU computation time: " << avgGpuComputeTime << " ms\n";
  std::cout << "Average GPU to CPU transfer time: " << avgGpuToCpuTime << " ms\n";
  std::cout << "Average CPU binary search time: " << avgCpuComputeTime << " ms\n";

  float avgTotalTransferTime = avgCpuToGpuTime + avgGpuToCpuTime;
  float avgTotalComputeTime = avgGpuComputeTime + avgCpuComputeTime;
  float totalAvgTime = avgTotalTransferTime + avgTotalComputeTime;

  std::cout << "Total average memory transfer time (CPU ↔ GPU): " << avgTotalTransferTime << " ms ("
            << (avgTotalTransferTime / totalAvgTime * 100) << "% of total time)\n";
  std::cout << "Total average compute time (GPU + CPU): " << avgTotalComputeTime << " ms ("
            << (avgTotalComputeTime / totalAvgTime * 100) << "% of total time)\n";
  std::cout << "Total average execution time per iteration: " << totalAvgTime << " ms\n";

  // Cleanup
  checkCudaError(hipFree(deviceData), "Failed to free device memory");

  return 0;
}